
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>    //taille<2^20


__global__ void vecsumKernel(unsigned int *dVec, int size){
  printf("kerneled\n");
	int taille=blockDim.x/2;
	int id = blockIdx.x*blockDim.x+threadIdx.x;

	while(taille>=1 && id<=taille){
		dVec[id] += dVec[taille+id];
		taille = taille / 2;
		__syncthreads();
	}
  
}

void vecsum (unsigned int *vec, unsigned int *sum, int size){
     int bytes = size * sizeof(unsigned int);
     unsigned int *dVec;
     printf("size: %i\n", size);
     hipMalloc((void **) &dVec, bytes);
     hipMemcpy(dVec, vec, bytes, hipMemcpyHostToDevice);
     vecsumKernel <<< 1, size >>> (dVec, size);
     hipMemcpy(sum, dVec, sizeof(unsigned int), hipMemcpyDeviceToHost);
     hipFree(dVec);
}




int main(int argc, char **argv){
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
  int size;
  unsigned int *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  size = 1 << size;
  if (size >= (1 << 20)){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }
   vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
  for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i])); 
}
 
  unsigned int sum=0;
  vecsum(vec, &sum, size);
  printf("sum = %u\n", sum);
  fclose(f);
}

