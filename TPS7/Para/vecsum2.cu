
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>    //taille<2^20


__global__ void vecsumKernel(unsigned int *dVec, unsigned int *dSum, int size){
	int taille=1023/2;

	int bid = blockIdx.x;
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int tid = threadIdx.x;
		if(id<size){
	while(taille>=1&& tid <= taille){
		dVec[tid]+=dVec[taille+tid];
    taille/=2;
    __syncthreads();
    }
    if(tid==0){
        dSum[bid]=dVec[blockIdx.x*blockDim.x];
    }
		
	while(taille>=1 && id <= taille){
		dVec[id] += dVec[taille+id];
		taille = taille / 2;
		__syncthreads();
	}
    }
}

void vecsum (unsigned int *vec, unsigned int *sum, int size){
     int bsize = 1024;
     int gsize = (size+bsize-1)/bsize;
     int bytes = size * sizeof(unsigned int);
     unsigned int *dVec, *dSum;
     printf("gsize: %i\n", gsize);
     hipMalloc((void **) &dVec, bytes);
     hipMalloc((void **) &dSum, gsize*sizeof(unsigned int));
     hipMemcpy(dVec, vec, bytes, hipMemcpyHostToDevice);
     vecsumKernel <<< 1, size >>> (dVec, dSum, size);
     hipMemcpy(sum, dSum, sizeof(unsigned int), hipMemcpyDeviceToHost);
     hipFree(dVec); hipFree(dSum);
}




int main(int argc, char **argv){
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
  int size;
  unsigned int *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  size = 1 << size;
  if (size >= (1 << 20)){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }
   vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
  for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i])); 
}
 
  unsigned int sum=0;
  vecsum(vec, &sum, size);
  printf("sum = %u\n", sum);
  fclose(f);
}

