#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <assert.h>    //taille<2^20


__global__ void vecsumKernel(unsigned int *dVec, unsigned int *dSum, int size){
	int taille=1023/2;
	int bid = blockIdx.x;
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	int tid = threadIdx.x;
	
	while(taille>=1&& tid <= taille){
		
		
	while(taille>=1 && id <= taille){
		dVec[id] += dVec[taille+id];
		taille = taille / 2;
		__syncthreads();
	}
}

void vecsum (unsigned int *vec, unsigned int *sum, int size){
     int bsize = 1024;
     int bytes = size * sizeof(unsigned int);
     int gsize = (size+bsize-1)/bsize;
     unsigned int *dVec;
     unsigned int *dSum;
     printf("size: %i\n", gsize);
     hipMalloc((void **) &dVec, bytes);
     hipMalloc((void **) &dSum, gsize);
     hipMemcpy(dVec, vec, bytes, hipMemcpyHostToDevice);
     vecsumKernel <<< gsize, bsize >>> (dVec, dSum, size);
     hipMemcpy(sum, dSum, sizeof(unsigned int), hipMemcpyDeviceToHost);
     hipFree(dVec);
}




int main(int argc, char **argv){
  if (argc < 2){
    printf("Usage: <filename>\n");
    exit(-1);
  }
  int size;
  unsigned int *vec;
  FILE *f = fopen(argv[1],"r");
  fscanf(f,"%d\n",&size);
  size = 1 << size;
  if (size >= (1 << 20)){
    printf("Size (%u) is too large: size is limited to 2^20\n",size);
    exit(-1);
  }
   vec = (unsigned int *) malloc(size * sizeof(unsigned int)); assert(vec);
  for (int i=0; i<size; i++){
    fscanf(f, "%u\n",&(vec[i])); 
}
 
  unsigned int sum=0;
  vecsum(vec, &sum, size);
  printf("sum = %u\n", sum);
  fclose(f);
}

